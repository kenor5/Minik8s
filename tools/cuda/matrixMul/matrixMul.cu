
#include <hip/hip_runtime.h>
#include <stdio.h>

// 定义矩阵大小
#define N 32

// CUDA 核函数，用于矩阵乘法计算
__global__ void matrixMultiply(float* A, float* B, float* C)
{
    // 计算当前线程的全局索引
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // 执行矩阵乘法运算
    float sum = 0.0;
    for (int k = 0; k < N; ++k) {
        sum += A[row * N + k] * B[k * N + col];
    }

    // 将结果保存到矩阵 C
    C[row * N + col] = sum;
}

int main()
{
    // 分配主机上的矩阵内存
    float* h_A = (float*)malloc(N * N * sizeof(float));
    float* h_B = (float*)malloc(N * N * sizeof(float));
    float* h_C = (float*)malloc(N * N * sizeof(float));

    // 初始化矩阵 A 和 B
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
    }

    // 分配设备上的矩阵内存
    float* d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    // 将矩阵 A 和 B 从主机内存复制到设备内存
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // 定义 CUDA 核的网格和块大小
    dim3 gridSize(N / 16, N / 16);
    dim3 blockSize(16, 16);

    // 调用 CUDA 核函数进行矩阵乘法计算
    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C);

    // 将结果从设备内存复制到主机内存
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果矩阵的一部分
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // 释放内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
