
#include <hip/hip_runtime.h>
#include <stdio.h>

// 定义矩阵大小
#define N 32

// CUDA 核函数：矩阵加法
__global__ void matrixAdd(int *a, int *b, int *c)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        int index = row * N + col;
        c[index] = a[index] + b[index];
    }
}

int main()
{
    // 定义矩阵大小和字节数
    int numBytes = N * N * sizeof(int);

    // 分配主机内存
    int *h_a = (int *)malloc(numBytes);
    int *h_b = (int *)malloc(numBytes);
    int *h_c = (int *)malloc(numBytes);

    // 初始化矩阵数据
    for (int i = 0; i < N * N; i++)
    {
        h_a[i] = i;
        h_b[i] = i;
    }

    // 分配设备内存
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, numBytes);
    hipMalloc((void **)&d_b, numBytes);
    hipMalloc((void **)&d_c, numBytes);

    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_a, h_a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, numBytes, hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // 启动 CUDA 核函数进行矩阵加法运算
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // 将结果从设备内存复制到主机内存
    hipMemcpy(h_c, d_c, numBytes, hipMemcpyDeviceToHost);

    // 打印结果
    for (int i = 0; i < N * N; i++)
    {
        printf("%d ", h_c[i]);
        if ((i + 1) % N == 0)
            printf("\n");
    }

    // 释放内存
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
